#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#define HISTOGRAM_SIZE 256


__global__  void cudaCalcHisto(int *arr,int numElements,int* histogram) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

     __shared__ int sharedHisto[HISTOGRAM_SIZE];

    //init shared histogram
	if(threadIdx.x==0){
		int j;
		for(j=0; j<HISTOGRAM_SIZE; j++){
			sharedHisto[j]=0;
		}
	}
	__syncthreads();
	
    //check match between threads to blocks 
    if (i < numElements)
    	atomicAdd(&sharedHisto[arr[i]],1);
    __syncthreads();
   if(threadIdx.x < HISTOGRAM_SIZE)
    	atomicAdd(&histogram[i], sharedHisto[i]);
}

int* cudaAllocate(int numElements,int* data)
{
   hipError_t err = hipSuccess;

    size_t size = numElements * sizeof(float);
  

    // Allocate memory on GPU to copy the data from the host
    int *d_A;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_A, data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);   
    }

    return d_A;
}


int computeOnGPU(int *data,int* histogram, int numElements) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int* cudaHisto=cudaAllocate(HISTOGRAM_SIZE,&histogram[0]);
    int* hipArray=cudaAllocate(numElements,&data[0]);


    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    cudaCalcHisto<<<blocksPerGrid, threadsPerBlock>>>(hipArray, numElements,cudaHisto);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(histogram,cudaHisto,HISTOGRAM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(hipArray) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

     if (hipFree(cudaHisto) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return 0;
}

